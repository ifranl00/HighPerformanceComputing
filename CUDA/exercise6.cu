//Parallelization - several blocks - several threads + balance

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#include<sys/time.h>
#define N 100000
#define M 8 //Threads per block

__global__ void add(int *a, int *b, int *c, int n)
{
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        if(index<n)
                c[index] = a[index] + b[index];
}

void random (int *tab, int wym )
{
        int i;
        for(i=0;i<wym;i++)
                tab[i]=rand()%101;
}


int main(void) {
        int *a, *b, *c; // host copies of a, b, c
        int *d_a, *d_b, *d_c; // device copies of a, b, c
        int size = N * sizeof(int);
        int i;
        srand(time(NULL));
        // Allocate space for device copies of a, b, c
        hipMalloc((void **)&d_a, size);
        hipMalloc((void **)&d_b, size);
        hipMalloc((void **)&d_c, size);
        // Alloc space for host copies of a, b, c and setup input values
        a = (int *)malloc(size);
        long cputime,seconds,temp;
        struct timeval start2,end2;
        gettimeofday(&start2,NULL);
         random(a, N);
        b = (int *)malloc(size);
         random(b, N);
        gettimeofday(&end2,NULL);
        seconds = end2.tv_sec - start2.tv_sec;
        temp = end2.tv_usec - start2.tv_usec;
        cputime = ((seconds)*1000 + temp/1000.0)+0.5;
        printf("cpu time: %ld ms\n",cputime);
        c = (int *)malloc(size);
        // Copy inputs to device
        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
        // Launch add() kernel on GPU
        float gputime=0;
        hipEvent_t start,stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start,0);
        add<<<(N+M-1)/M,M>>>(d_a, d_b, d_c, N);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&gputime,start,stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        printf("GPU time: %.4f ms\n",gputime);
        // Copy result back to host
        hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

        for(i=0;i<N;i++)
        {
                //printf("a[%d](%d) + b[%d](%d) = c[%d](%d)\n",i,a[i],i,b[i],i,c[i]);
        }
        // Cleanup
        //printf("%d+%d=%d\n",a,b,c);
        free(a); free(b); free(c);
        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        printf("Total time: %lf ms\n",cputime+gputime);
        return 0;
}
